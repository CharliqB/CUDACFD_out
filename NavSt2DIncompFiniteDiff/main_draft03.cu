#include "hip/hip_runtime.h"
/* main.cu
 * \file main.cu
 * Navier-Stokes equation solver in 2-dimensions, incompressible flow, by finite difference
 * \author Ernest Yeung  
 * \email ernestyalumni@gmail.com
 * \date 20161206
 * 
 * Compilation tips if you're not using a make file
 * 
 * nvcc -std=c++11 -c ./physlib/R2grid.cpp -o R2grid.o  // or 
 * g++ -std=c++11 -c ./physlib/R2grid.cpp -o R2grid.o
 * 
 * nvcc -std=c++11 -c ./physlib/dev_R2grid.cu -o dev_R2grid.o
 * nvcc -std=c++11 main.cu R2grid.o dev_R2grid.o o main.exe
 * 
 */
/*
 * cf. Kyle e. Niemeyer, Chih-Jen Sung.  
 * Accelerating reactive-flow simulations using graphics processing units.  
 * AIAA 2013-0371  American Institute of Aeronautics and Astronautics.  
 * http://dx.doi.org/10.5281/zenodo.44333
 * 
 * Michael Griebel, Thomas Dornsheifer, Tilman Neunhoeffer. 
 * Numerical Simulation in Fluid Dynamics: A Practical Introduction (Monographs on Mathematical Modeling and Computation). 
 * SIAM: Society for Industrial and Applied Mathematics (December 1997). 
 * ISBN-13:978-0898713985 QA911.G718 1997
 * 
 * */ 

#include <iomanip>					// std::setprecision
#include <iostream> 				// std::cout
#include <cmath>    				// std::sqrt, std::fmax 

#include "./physlib/R2grid.h"      	// Grid2d
#include "./physlib/dev_R2grid.h"  	// Dev_Grid2d
#include "./physlib/u_p.h"          // compute_F, compute_G, compute_RHS, etc.
#include "./physlib/boundary.h"     // set_BConditions_host, set_BConditions, set_lidcavity_BConditions_host, set_lidcavity_BConditions
#include "./commonlib/checkerror.h" // checkCudaErrors

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>  // thrust::max_element (thrust::min_element)  
#include <thrust/copy.h>  // thrust::copy
#include <thrust/execution_policy.h> // thrust::device

// File I/O includes
#include <fstream> 		// std::ofstream

#include <array> 		// std::array

int main(int argc, char* argv[]) {
	// ################################################################
	// ####################### Initialization #########################
	// ################################################################
	
	// discretization (parameters) <==> graphical (parameters)
	const int L_X { 512 };  			// WIDTH   // I've tried values 32.  128, 32, 0.5 works; 256, 32, 0.25 works (even though 256, 64 doesn't); 512, 64, doesn't work, neither does 512,32; 512, 16 works
	const int L_Y { 512 };  			// HEIGHT  // I've tried values 32,  128, 32, 0.5 works

	// "real", physical parameters
	/** try domain size (non-dimensional) */
	constexpr const float l_X = 1.0;  	// length (in x-direction)
	constexpr const float l_Y = 1.0; 	// height (in y-direction)

	// physics (on device); Euclidean (spatial) space
	dim3 dev_L2 { static_cast<unsigned int>(L_X), 
				static_cast<unsigned int>(L_Y) };

	Dev_Grid2d dev_grid2d( dev_L2); 

	// physics (on host); Euclidean (spatial) space
	constexpr std::array<int,2> LdS { L_X, L_Y } ;
	constexpr std::array<float,2> ldS { l_X, l_Y };

	Grid2d grid2d{LdS, ldS};

	// dynamics (parameters)
	const dim3 M_i { 32, 32 }; 	// number of threads per block, i.e. Niemeyer's BLOCK_SIZE // I've tried values 4,4

	float t = 0.0 ;
	int cycle = 0;
	
	// iterations for SOR successive over relaxation
	int iter = 0;
	int itermax = 10000;  // I tried values such as 10000, Griebel, et. al. = 100

	/* READ the parameters of the problem                 */
	/* -------------------------------------------------- */ 

	/** Safety factor for time step modification; safety factor for time stepsize control */
	constexpr const float tau = 0.5; 

	/** SOR relaxation parameter; omg is Griebel's notation */
	constexpr const float omega = 1.7;  
	
	/** Discretization mixture parameter (gamma); gamma:upwind differencing factor is Griebel's notation */
	constexpr const float gamma = 0.9;

	/** Reynolds number */
	constexpr const float Re_num = 1000.0;

	// SOR iteration tolerance
	const float tol = 0.001;  // Griebel, et. al., and Niemeyer has this at 0.001
	
	// time range
	const float time_start = 0.0;
	const float time_end = 1.4000;  // L_X=L_Y=128, M_i=32, t_f=0.5 works
	
	// initial time step size
	float deltat = 0.02; // I've tried values 0.002
	
	// set initial BCs on host CPU
	set_BConditions_host( grid2d );
	set_lidcavity_BConditions_host( grid2d );

	set_BConditions( dev_grid2d );
	set_lidcavity_BConditions( dev_grid2d );

	/* delt satisfying CFL conditions */
	/* ------------------------------ */
	float max_u = 1.0e-10;
	float max_v = 1.0e-10;

	// variables to store maximum velocities
	thrust::device_vector<float> max_u_vec(grid2d.NFLAT());
	thrust::device_vector<float> max_v_vec(grid2d.NFLAT());

	// get max velocity for initial values (including BCs)
	#pragma unroll
	for (auto j = 1; j < grid2d.staggered_Ld[1]; ++j) {
		#pragma unroll
		for (auto i = 0; i < grid2d.staggered_Ld[0]; ++i) {
			max_u = std::fmax( max_u, 
				fabs( dev_grid2d.u[ i + grid2d.staggered_Ld[0] * j] )  ) ;
		}
	}	
	#pragma unroll
	for (auto j = 0; j < grid2d.staggered_Ld[1]; ++j) {
		#pragma unroll
		for (auto i = 1; i < grid2d.staggered_Ld[0]; ++i) {
			max_v = std::fmax( max_v, 
				fabs( dev_grid2d.v[ i + grid2d.staggered_Ld[0] * j] )  ) ;
		}
	}	
	

// This is why you can't do (dev_grid2d->u).end()
// cf. http://stackoverflow.com/questions/13104138/error-expression-must-have-a-pointer-type-when-using-the-this-keyword
/*	thrust::device_vector<float>::iterator max_u_iter = 
		thrust::max_element( dev_grid2d.u.begin(), dev_grid2d.u.end() );
	max_u = std::fmax( *max_u_iter, max_u ) ;

	thrust::device_vector<float>::iterator max_v_iter = 
		thrust::max_element( dev_grid2d.v.begin(), dev_grid2d.v.end() );
	max_v = std::fmax( *max_v_iter, max_v ) ;
*/	

	////////////////////////////////////////	
	// block and grid dimensions
	// "default" gridSize is number of blocks on a grid along a dimension
	dim3 gridSize ( (grid2d.staggered_Ld[0] + M_i.x -1)/M_i.x, 
						(grid2d.staggered_Ld[1] + M_i.y - 1)/M_i.y) ;
	
	// pressure kernel, alternative way to launch threads in blocks, so called sPencil
/*	dim3 block_press( M_i.x, 1 ) ; 
	dim3 grid_press( ( grid2d.staggered_Ld[0] + block_press.x - 1)/block_press.x , grid2d.staggered_Ld[1] ) ;
	* */
	////////////////////////////////////////

	// residual variable
	// residualsquared thrust device vector
	thrust::device_vector<float> residualsq(grid2d.staggered_SIZE() );
	float* residualsq_Array = thrust::raw_pointer_cast( residualsq.data() );

	
	// pressure sum 
	/* Note that the pressure summation needed to normalize to the pressure magnitude for 
	 * relative tolerance is, in Griebel, et. al's implementation, the first part of the 
	 * POISSON routine, and used at the very end of POISSON, here in the GPU implementation
	 * it's separated */ 
	thrust::device_vector<float> pres_sum_vec(grid2d.NFLAT());
	float* pres_sum_Arr = thrust::raw_pointer_cast( pres_sum_vec.data() );
	

	// time-step size based on grid and Reynolds number
	float dt_Re = 0.5 * Re_num / ((1.0 / (grid2d.hd[0] * grid2d.hd[0])) + (1.0 / (grid2d.hd[1] * grid2d.hd[1])));
	
	// ################################################################
	// #######################               ##########################
	// #######################   MAIN LOOP   ##########################
	// #######################               ##########################
	// ################################################################
	
				/* t i m e    l o o p */
				/* ------------------ */
	// time iteration loop
	for (t=time_start,cycle=0; t < time_end; cycle++) {

		// calculate time step based on stability and CFL
		deltat = std::fmin( (grid2d.hd[0] / max_u), ( grid2d.hd[1]/ max_v) );
		deltat = tau * std::fmin( dt_Re, deltat);

		if ((t+deltat) >= time_end) {
			deltat = time_end - t; }
	
	
		/* Compute tentative velocity field (F,G) */
		// i.e. calculate F and G
		/* -------------------------------------- */			
		compute_F<<<gridSize,M_i>>>( deltat, 
			dev_grid2d.u_arr, dev_grid2d.v_arr, dev_grid2d.F_arr,
			grid2d.Ld[0], grid2d.Ld[1], grid2d.hd[0], grid2d.hd[1],
			gamma, Re_num); 

		compute_G<<<gridSize,M_i>>>( deltat, 
			dev_grid2d.u_arr, dev_grid2d.v_arr, dev_grid2d.G_arr,
			grid2d.Ld[0], grid2d.Ld[1], grid2d.hd[0], grid2d.hd[1],
			gamma, Re_num); 

	

	// get L2 norm of initial pressure
	for (auto j = 0; j < dev_grid2d.staggered_Ld.y; ++j) {
		for (auto i = 0; i < dev_grid2d.staggered_Ld.x; ++i) {
			if ((i>0)&&(i<(dev_grid2d.Ld.x+1)) && (j>0) && (j<(dev_grid2d.Ld.y+1))) {
				int k = (i-1) + dev_grid2d.Ld.x * (j-1) ; 
				pres_sum_vec[k] = dev_grid2d.p[ dev_grid2d.staggered_flatten(i,j) ] ; 
			}
		}
	}
	float p0_norm = 0.0;
	p0_norm = thrust::reduce( pres_sum_vec.begin(), pres_sum_vec.end(), 0, thrust::plus<float>() );
	
	p0_norm =sqrt(p0_norm / (static_cast<float>( grid2d.NFLAT() ) ));
	
	if (p0_norm < 0.0001) {
		p0_norm = 1.0;
	}
	
	// ensure all kernels are finished
	hipDeviceSynchronize();
	
	/* Compute right hand side for pressure equation */
	/* --------------------------------------------- */
	compute_RHS<<<gridSize,M_i>>>( dev_grid2d.F_arr, dev_grid2d.G_arr,
		dev_grid2d.RHS_arr, 
		dev_grid2d.Ld.x, dev_grid2d.Ld.y, 
		deltat, grid2d.hd[0], grid2d.hd[1] );
	
	float norm_L2; // residual; res for Griebel, et. al., norm_L2 for Niemeyer
	
	/* Solve the pressure equation by successive over relaxation */
	/* ---------------------------------------------------------- */
	// calculate new pressure
	for (iter = 1; iter <= itermax; iter++) {
		// set pressure boundary conditions
		set_horiz_press_BCs( dev_grid2d ) ;
		set_vert_press_BCs( dev_grid2d ) ;
	
		// ensure kernel finished
		hipDeviceSynchronize();
	
		// operations needed to do poisson; poisson and thrust::swap
/*
		poisson<<<gridSize, M_i>>>( dev_grid2d.p_arr, dev_grid2d.RHS_arr, 
			dev_grid2d.p_temp_arr, 
			grid2d.Ld[0], grid2d.Ld[1], grid2d.hd[0], grid2d.hd[1], omega) ; 

		thrust::copy( thrust::device, dev_grid2d.p_temp.begin(), dev_grid2d.p_temp.end(), dev_grid2d.p.begin() );
			// dev_grid2d.p is no a copy of dev_grid2d.p_temp
*/
		// END of operations needed to do poisson; poisson and thrust::swap


		
		poisson_redblack<<<gridSize, M_i>>>( dev_grid2d.p_arr, dev_grid2d.RHS_arr, 
			grid2d.Ld[0], grid2d.Ld[1], grid2d.hd[0], grid2d.hd[1], omega) ; 


		// calculate residual values
		compute_residual<<<gridSize, M_i>>>( dev_grid2d.p_arr, dev_grid2d.RHS_arr, 
			grid2d.Ld[0], grid2d.Ld[1], grid2d.hd[0], grid2d.hd[1], 
			residualsq_Array) ; 

		
		norm_L2 = thrust::reduce( residualsq.begin(), residualsq.end(), 0, thrust::plus<float>() );
		
		// calculate residual
		norm_L2 = sqrt( norm_L2/ ( static_cast<float>( grid2d.NFLAT() )) ) / p0_norm;


		// if tolerance has been reached, end SOR iterations
		if (norm_L2 < tol) {
			break;
		}				
	} // END for loop, to solve the pressure equation by SOR

	std::cout << "Time = " << t + deltat << ", delta t = " << deltat << ", iter = " 
		<< iter << 	", res (or norm_L2) = " << std::setprecision(9) << norm_L2 << 
		", max_u : " << max_u << ", max_v : " << max_v << ", p0_norm : " << p0_norm << 
		", cycle = " << cycle << std::endl; 

		/* Compute the new velocity field */
		// i.e. calculate new velocities
		/* ------------------------------ */

		calculate_u<<<gridSize,M_i>>>( dev_grid2d.u_arr, dev_grid2d.p_arr,
			dev_grid2d.F_arr, grid2d.Ld[0], grid2d.Ld[1], deltat, grid2d.hd[0] );

		calculate_v<<<gridSize,M_i>>>( dev_grid2d.v_arr, dev_grid2d.p_arr,
			dev_grid2d.G_arr, grid2d.Ld[0], grid2d.Ld[1], deltat, grid2d.hd[1] );


		// get maximum u- and v- velocities
		max_v = 1.0e-10;
		max_u = 1.0e-10;
	
		for (auto j = 0 ; j < grid2d.Ld[1] ; ++j) { 
			for (auto i = 0; i < grid2d.Ld[0]; ++i) {
				max_u_vec[i + grid2d.Ld[0]*j] = dev_grid2d.u[ (i+1) + grid2d.staggered_Ld[0] * (j+1) ] ;  
				max_v_vec[i + grid2d.Ld[0]*j] = dev_grid2d.v[ (i+1) + grid2d.staggered_Ld[0] * (j+1) ] ;
			}
		}

		thrust::device_vector<float>::iterator max_u_iter = 
			thrust::max_element( max_u_vec.begin(), max_u_vec.end() );
		max_u = std::fmax( *max_u_iter, max_u);

		thrust::device_vector<float>::iterator max_v_iter = 
			thrust::max_element( max_v_vec.begin(), max_v_vec.end() );
		max_v = std::fmax( *max_v_iter, max_v);



		// set velocity boundary conditions
		/* Set boundary conditions */
		/* ----------------------- */
		set_BConditions( dev_grid2d ) ;
		
		/* Set special boundary conditions */
		/* Overwrite preset default values */
		/* ------------------------------- */
		set_lidcavity_BConditions( dev_grid2d  );

		hipDeviceSynchronize();

		// increase time
		t += deltat;

	} // END end for loop, time iteration loop 

	// sanity check
	std::cout << " Right after time iteration loop, final p : " << std::endl;
	std::cout << "\n dev_grid2d.p : " << std::endl; 
//		for (auto j = (grid2d.staggered_Ld[1]-1); j >= 0; --j) {
//			for (auto i = 0; i < grid2d.staggered_Ld[0]; ++i) {
		for (auto j = (grid2d.staggered_Ld[1]-1)/2; j >= 0; --j) {
			for (auto i = 0; i < grid2d.staggered_Ld[0]/2; ++i) {
				std::cout << std::setprecision(3) << dev_grid2d.p[i+(grid2d.staggered_Ld[0])*j] << " " ; }
			std::cout << std::endl ; }
	// END of sanity check

	// ################################################################
	// ########################## File I/O ############################
	// ################################################################

	std::ofstream ofile; // no file mode is set
	ofile.open("velocity_gpu.dat"); // mode implicitly out and trunc

	ofile << "#x\ty\tu\tv\n" ;
	
	std::array<int,2> ix_ind;
	std::array<float,2> x_ind;
	
	for (auto j = 0; j <= grid2d.Ld[1]; j++) {
		for (auto i = 0; i <= grid2d.Ld[0]; i++) {
			float u_ij = dev_grid2d.u[ i + grid2d.staggered_Ld[0] * j ] ; 
			float v_ij = dev_grid2d.v[ i + grid2d.staggered_Ld[0] * j ] ; 
			
			ix_ind[0] = i; 
			ix_ind[1] = j; 
			x_ind = grid2d.gridpt_to_space( ix_ind ) ;
			
			ofile << x_ind[0] << "\t" << x_ind[1] << "\t" << u_ij << "\t" << v_ij << std::endl;
		}
	}
	ofile.close();

	
	std::cout << " End of program " << std::endl;
	return 0;
} 

